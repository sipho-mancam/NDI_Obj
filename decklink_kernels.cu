#include "hip/hip_runtime.h"

#include "decklink_kernels.cuh"
#include <opencv2/opencv.hpp>


// unpack to half ....
__global__ void unpack_10bit_yuv_h(uint4* source, uint4* dst, size_t width, size_t height)
{
	// width is the original image width ... 1920 or 3840 

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int srcWidth = width / 6, dstWidth = width / 2; 


	uint4* macroPx;
	macroPx = &source[y * srcWidth + x];

	double Cr0;
	double Y0;

	double Cb0;
	double Y1;

	double Cb2;
	double Y2;

	double Cr2;
	double Y3;

	double Cb4;
	double Y5;

	double Cr4;
	double Y4;

	Cb0 = (macroPx->x & 0x3ff);
	Y0 = ((macroPx->x & 0xffc00) >> 10);

	Cr0 = (macroPx->x >> 20);
	Y1 = (macroPx->y & 0x3ff);
	
	Cb2 = ((macroPx->y & 0xffc00) >> 10);
	Y2 = (macroPx->y >> 20);

	Cr2 = (macroPx->z & 0x3ff);
	Y3 = ((macroPx->z & 0xffc00) >> 10);
	
	Cb4 = (macroPx->z >> 20);
	Y4 = (macroPx->w & 0x3ff);

	Cr4 = ((macroPx->w & 0xffc00) >> 10);
	Y5 = (macroPx->w >> 20);
	
	dst[y * dstWidth + (x * 3) + 0] = make_uint4(Cr0, Y0, Cb0, Y1); // x y z w
	dst[y * dstWidth + (x * 3) + 1] = make_uint4(Cr2, Y2, Cb2, Y3);
	dst[y * dstWidth + (x * 3) + 2] = make_uint4(Cr4, Y4, Cb4, Y5);
}

__global__ void unpack_10bit_yuv_f(uint4* source, uint* dst, size_t width, size_t height)
{
	// width is the original image width ... 1920 or 3840 
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	int srcWidth = width / 6;

	if (x >= srcWidth || y >= height)
		return;

	uint4* macroPx;
	macroPx = &source[y * srcWidth + x];

	double Cr0;
	double Y0;

	double Cb0;
	double Y1;

	double Cb2;
	double Y2;

	double Cr2;
	double Y3;

	double Cb4;
	double Y5;

	double Cr4;
	double Y4;

	Cb0 = (macroPx->x & 0x3ff);
	Y0 = ((macroPx->x & 0xffc00) >> 10);

	Cr0 = (macroPx->x >> 20);
	Y1 = (macroPx->y & 0x3ff);

	Cb2 = ((macroPx->y & 0xffc00) >> 10);
	Y2 = (macroPx->y >> 20);

	Cr2 = (macroPx->z & 0x3ff);
	Y3 = ((macroPx->z & 0xffc00) >> 10);

	Cb4 = (macroPx->z >> 20);
	Y4 = (macroPx->w & 0x3ff);

	Cr4 = ((macroPx->w & 0xffc00) >> 10);
	Y5 = (macroPx->w >> 20);


	dst[y * width + (x * 18) + 0] = Cb0;
	dst[y * width + (x * 18) + 1] = 0;
	dst[y * width + (x * 18) + 2] = Y0;

	dst[y * width + (x * 18) + 3] = 0;
	dst[y * width + (x * 18) + 4] = Cr0;
	dst[y * width + (x * 18) + 5] = Y1;

	dst[y * width + (x * 18) + 6] = Cb2;
	dst[y * width + (x * 18) + 7] = 0;
	dst[y * width + (x * 18) + 8] = Y2;

	dst[y * width + (x * 18) + 9] = 0;
	dst[y * width + (x * 18) + 10] = Cr2;
	dst[y * width + (x * 18) + 11] = Y3;

	dst[y * width + (x * 18) + 12] = Cb4;
	dst[y * width + (x * 18) + 13] = 0;
	dst[y * width + (x * 18) + 14] = Y4;

	dst[y * width + (x * 18) + 15] = 0;
	dst[y * width + (x * 18) + 16] = Cr4;
	dst[y * width + (x * 18) + 17] = Y5;
}


__global__ void unpack_10bit_rbg(uchar* source, uint4* dst, size_t width, size_t height)
{

}

__global__ void pack_10bit_yuv(uint4* source, uint4* dst, size_t width, size_t height)
{

}

// pack 10bit yuv --> yu yv --> to pixels per word
__global__ void pack_8bit_yuv(uchar* source, uint* dst, size_t width, size_t height)
{

}

void alpha_2_decklink(long width, long height, uchar *alpha_channel /*Host buffer*/, uint** output)
{

	hipError_t cudaStatus;
	const dim3 block(16, 16); // 256 threads per block..

	int rows = height / block.y;
	if (height % block.y > 0)
		rows += block.y - height % block.y;

	const dim3 grid(width / (2 * block.x), rows);

	uchar* pinnedBuf, *in_gpuBuf;
	uint* gpuBuf_out;
	uint* cpuOut;

	size_t packedSize = (width / 2) * height * sizeof(uint);

	assert(hipSuccess == hipHostMalloc((void**)&pinnedBuf, width * height*sizeof(uchar)));
	assert(hipSuccess == hipHostMalloc((void**)&cpuOut, packedSize));

	memcpy(pinnedBuf, alpha_channel, width*height); // single channel copy

	assert(hipSuccess == hipMalloc((void**)&in_gpuBuf, width * height));
	assert(hipSuccess == hipMemcpy((void*)in_gpuBuf, (void*)pinnedBuf, width * height, hipMemcpyHostToDevice));
	assert(hipMalloc((void**)&gpuBuf_out, packedSize) == hipSuccess);
	
	alpha_2_yuyv_pack <<< grid, block >>> (
		in_gpuBuf,
		gpuBuf_out,
		width, height
		);

	cudaStatus = hipGetLastError();
	assert(cudaStatus == hipSuccess);
	assert(hipSuccess == hipDeviceSynchronize());

	cudaStatus = hipMemcpy(cpuOut, gpuBuf_out, packedSize, hipMemcpyDeviceToHost);
	assert(cudaStatus == hipSuccess);
	*output = cpuOut;

	hipHostFree(pinnedBuf);
	hipFree(in_gpuBuf);
	hipFree(gpuBuf_out);
}


void alpha_2_decklink_gpu(long width, long height, uchar* alpha_channel /*GPU Buffer*/, uint** output /*CPU Buffer*/)
{

	hipError_t cudaStatus;
	const dim3 block(16, 16); // 256 threads per block..
	const dim3 grid(width / (2 * block.x), round(height+8 / block.y));

	uint* gpuBuf_out;
	uint* cpuOut;

	size_t packedSize = (width / 2) * height * sizeof(uint);

	assert(hipSuccess == hipHostMalloc((void**)&cpuOut, packedSize));
	assert(hipMalloc((void**)&gpuBuf_out, packedSize) == hipSuccess);

	alpha_2_yuyv_pack <<< grid, block >>> (
		alpha_channel,
		gpuBuf_out,
		width, height
		);

	cudaStatus = hipGetLastError();
	assert(cudaStatus == hipSuccess);
	assert(hipSuccess == hipDeviceSynchronize());

	cudaStatus = hipMemcpy(cpuOut, gpuBuf_out, packedSize, hipMemcpyDeviceToHost);
	assert(cudaStatus == hipSuccess);

	*output = cpuOut;

	hipFree(alpha_channel);
	hipFree(gpuBuf_out);
}

__global__ void alpha_2_yuyv_pack(uchar* source, uint* dst, size_t width, size_t height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int dstWidth = width / 2;

	uchar Y0, Y1, Cb, Cr;

	Y0 = source[y * width + (x * 2)];
	Y1 = source[y * width + (x * 2) + 1];
	Cb = -127;
	Cr = -127;

	dst[y * dstWidth + x] |= (uint)(Y1 << 24);
	dst[y * dstWidth + x] |= (uint)(Cr << 16);
	dst[y * dstWidth + x] |= (uint)(Y0 << 8);
	dst[y * dstWidth + x] |= (uint)(Cb);

}

void get_alpha_channel(long width, long height, uchar* bgra, uchar** alpha_out)
{
	hipError_t cudaStatus;
	const dim3 block(16, 16); // 256 threads per block..
	const dim3 grid(width / block.x, height / block.y);

	uchar * in_gpu_buf; // bgra pinned and gpu buffers
	uchar* pinned_alpha, *out_alpha;

	size_t bgra_size = width * height * 4;
	size_t alpha_size = width * height * 1;

	assert(hipSuccess == hipMalloc((void**)&in_gpu_buf, bgra_size));
	assert(hipSuccess == hipMemcpy(in_gpu_buf, bgra, bgra_size, hipMemcpyHostToDevice));
	// BGRA data is now in device memory. ...
	assert(hipSuccess == hipMalloc((void**)&out_alpha, alpha_size));

	bgra_2_alpha <<<grid, block >>> (
		in_gpu_buf,
		out_alpha,
		width, height
		);

	cudaStatus = hipGetLastError();
	assert(cudaStatus == hipSuccess);
	assert(hipSuccess == hipDeviceSynchronize());

	assert(hipSuccess == hipHostMalloc((void**)&pinned_alpha, alpha_size));
	assert(hipSuccess == hipMemcpy(pinned_alpha, out_alpha, alpha_size, hipMemcpyDeviceToHost));

	*alpha_out = pinned_alpha;

	hipFree(in_gpu_buf);
	hipFree(out_alpha);
}


void get_alpha_channel_gpu(long width, long height, uchar* bgra /*GPU buffer*/, uchar** alpha_out)
{
	hipError_t cudaStatus;
	const dim3 block(16, 16); // 256 threads per block..
	const dim3 grid(width / block.x, height / block.y);

	uchar * out_alpha;
	size_t alpha_size = width * height;

	assert(hipSuccess == hipMalloc((void**)&out_alpha, alpha_size));

	bgra_2_alpha << <grid, block >> > (
		bgra,
		out_alpha,
		width, height
		);

	cudaStatus = hipGetLastError();
	assert(cudaStatus == hipSuccess);
	assert(hipSuccess == hipDeviceSynchronize());

	*alpha_out = out_alpha;

	//hipFree(in_gpu_buf);
	hipFree(bgra);

}

__global__ void bgra_2_alpha(uchar* bgra, uchar* alpha, long width, long height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int numChannels = 4;
	int selectedChannel = 3;

	alpha[y * width + x] = bgra[y * (width * numChannels) + (x * numChannels)+ selectedChannel];
}

uchar* get_yuv_from_bgr_packed(long width, long height, uchar* bgra, uint** output)
{
	hipError_t cudaStatus;
	const dim3 block(16, 16); // 256 threads per block..
	const dim3 grid(width / (2*block.x), height / block.y);

	uchar* in_gpu_buf; // bgra pinned and gpu buffers
	uint* pinned_yuv, * out_yuv;

	size_t bgra_size = width * height * 4;
	size_t yuv_size = sizeof(uint) * (width / 2) * height;

	assert(hipSuccess == hipMalloc((void**)&in_gpu_buf, bgra_size));
	assert(hipSuccess == hipMemcpy(in_gpu_buf, bgra, bgra_size, hipMemcpyHostToDevice));
	// BGRA data is now in device memory. ...
	assert(hipSuccess == hipMalloc((void**)&out_yuv, yuv_size));
	assert(hipSuccess == hipHostMalloc((void**)&pinned_yuv, yuv_size));


	bgr_2_8bityuv_packed << <grid, block >> > (
		in_gpu_buf,
		out_yuv,
		width, height
		);

	cudaStatus = hipGetLastError();
	assert(cudaStatus == hipSuccess);
	assert(hipSuccess == hipDeviceSynchronize());

	assert(hipSuccess == hipMemcpy(pinned_yuv, out_yuv, yuv_size, hipMemcpyDeviceToHost));
	*output = pinned_yuv;

	hipFree(out_yuv);

	return in_gpu_buf;

}


__global__ void bgr_2_8bityuv_packed(uchar* bgra, uint* dst, long width, long height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int dstWidth = width / 2;
	int numChannels = 4;

	uchar Y0, Y1, Cb, Cr;

	uchar R1, R2, G1, G2, B1, B2;

	B1 = bgra[y * (width * numChannels) + ((2 * x) * numChannels) + 0];
	G1 = bgra[y * (width * numChannels) + ((2 * x) * numChannels) + 1];
	R1 = bgra[y * (width * numChannels) + ((2 * x) * numChannels) + 2];


	B2 = bgra[y * (width * numChannels) + (((2 * x) + 1) * numChannels) + 0];
	G2 = bgra[y * (width * numChannels) + (((2 * x) + 1) * numChannels) + 1];
	R2 = bgra[y * (width * numChannels) + (((2 * x) + 1) * numChannels) + 2];


	// convert to YUV color space ....
	Y0 = 0.299 * R1 + 0.587 * G1 + 0.114 * B1;
	Cb = -1 * 0.299 * R1 - 0.587 * G1 + 0.886 * B1;

	Y1 = 0.299 * R2 + 0.587 * G2 + 0.114 * B2;
	Cr = 0.701 * R1 - 0.587 * G1 - 0.114 * B1;

	// pack it for decklink ....
	dst[y * dstWidth + x] |= (uint)(Y1 << 24);
	dst[y * dstWidth + x] |= (uint)(Cr << 16);
	dst[y * dstWidth + x] |= (uint)(Y0 << 8);
	dst[y * dstWidth + x] |= (uint)(Cb);

}
