#include "hip/hip_runtime.h"
#include "decklink_api.hpp"

#include "hip/hip_runtime_api.h"
#include ""
#include "decklink_kernels.hpp"

void VideoFrameCallback::unpack_10bit_yuv()
{
    hipError_t cudaStatus;
    const dim3 block(16, 16); // 256 threads per block..
    const dim3 grid(width / (6 * block.x), height / block.y);

    unpack_10bit_yuv_h <<< grid, block >>> (
        (uint4*)gpuMemory,
        dst_4,
        width, height
        );

    cudaStatus = hipGetLastError();
    assert(hipSuccess == cudaStatus);
    cudaStatus = hipDeviceSynchronize();
    assert(hipSuccess == cudaStatus);
}

void VideoFrameCallback::convert_10bit_2_rgb()
{
    hipError_t cudaStatus;
    const dim3 block(16, 16); // 256 threads per block..
    const dim3 grid(width /(2*block.x), height / block.y);
    unpacked_10bityuv_2_rgb <<< grid, block >> > (
        dst_4,
        rgb_data,
        width, height
        );

  

    cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
    assert(hipSuccess == hipDeviceSynchronize());
    assert(hipSuccess == hipMemcpy(rgb_data_h, rgb_data, width * height * sizeof(uchar3), hipMemcpyDeviceToHost));



}
