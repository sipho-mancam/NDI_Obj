#include "hip/hip_runtime.h"
#include "decklinkAPI.hpp"
#include "hip/hip_runtime_api.h"
#include ""
#include "decklink_kernels.cuh"

void VideoFrameCallback::unpack_10bit_yuv()
{
    hipError_t cudaStatus;
    const dim3 block(16, 16); // 256 threads per block..
    const dim3 grid(width / (6 * block.x), height / block.y);

    unpack_10bit_yuv_h <<< grid, block >>> (
        (uint4*)gpuMemory,
        dst_4,
        width, height
        );

    cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
    assert(hipSuccess == hipDeviceSynchronize());
}

